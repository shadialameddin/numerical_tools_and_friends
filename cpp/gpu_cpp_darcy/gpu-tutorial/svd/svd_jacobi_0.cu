
// C++-ified SVD example from
// https://docs.nvidia.com/cuda/cusolver/index.html#svd_examples

#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <hipsolver.h>

#include <cassert>
#include <iostream>
#include <vector>

inline void printMatrix(int m, int n, const double* A, int rows_A, const char* name)
{
    for (int row = 0; row < m; row++)
    {
        for (int col = 0; col < n; col++)
        {
            std::cout << name << "(" << row + 1 << ", " << col + 1
                      << ") = " << A[row + col * rows_A] << '\n';
        }
    }
}

inline void check(hipError_t&& error)
{
    if (error != hipSuccess)
    {
        throw std::domain_error("CUDA error");
    }
}

inline void check(hipblasStatus_t&& status)
{
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        throw std::domain_error("Cuda BLAS error");
    }
}

inline void check(hipsolverStatus_t&& status)
{
    if (status != HIPSOLVER_STATUS_SUCCESS)
    {
        throw std::domain_error("Cuda SOLVER error");
    }
}

inline void check(hiprandStatus_t&& status)
{
    if (status != HIPRAND_STATUS_SUCCESS)
    {
        throw std::domain_error("Cuda RAND error");
    }
}

class gpu_matrix
{
public:
    gpu_matrix(int64_t rows, int64_t columns)
    {
        m = rows;
        n = columns;
        lda = rows;
        check(hipMalloc((void**)&device_matrix, sizeof(double) * m * n));
    };
    auto data() { return device_matrix; };
    auto rows() { return m; };
    auto columns() { return n; };
    auto leading_dimension() { return lda; };

private:
    double* device_matrix = nullptr;
    int64_t m;
    int64_t n;
    int64_t lda;
};

// compute C = alpha * A * B + beta * C (double real matrices)
void gpu_matrix_multiplication(hipblasHandle_t cublas_handle,
                               gpu_matrix A,
                               bool A_is_transposed,
                               gpu_matrix B,
                               bool B_is_transposed,
                               gpu_matrix C,
                               const double alpha = 1,
                               const double beta = 0)
{
    check(cublasDgemm(cublas_handle,
                      (A_is_transposed) ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                      (B_is_transposed) ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                      A.rows(),
                      B.columns(),
                      A.rows(),
                      &alpha,
                      A.data(),
                      A.leading_dimension(),
                      B.data(),
                      B.leading_dimension(),
                      &beta,
                      C.data(),
                      C.leading_dimension()));
}

int main(int argc, char* argv[])
{
    hipsolverHandle_t cusolverH = nullptr;
    hipblasHandle_t cublasH = nullptr;
    hipStream_t stream = nullptr;
    hipsolverGesvdjInfo_t gesvdj_params = nullptr;

    const int rows_A = 3;
    const int cols_A = 2;
    const int modes = std::min(cols_A, 2);

    ///       | 1 2  |
    ///   A = | 4 5  |
    ///       | 2 1  |
    std::vector<double> A = {1.0, 4.0, 2.0, 2.0, 5.0, 1.0};
    // m-by-m unitary matrix, left singular vectors
    std::vector<double> U(rows_A * rows_A);
    // n-by-n unitary matrix, right singular vectors
    std::vector<double> V(cols_A * cols_A);
    // numerical singular value
    std::vector<double> S(cols_A);
    // exact singular values
    std::vector<double> const S_exact = {7.065283497082729, 1.040081297712078};

    // device copy of A
    double* d_A = nullptr;
    // singular values
    double* d_S = nullptr;
    // left singular vectors
    double* d_U = nullptr;
    // right singular vectors
    double* d_V = nullptr;
    // error info
    int* d_info = nullptr;

    // devie workspace for gesvdj
    double* d_work = nullptr;
    // host copy of error info
    int info = 0;

    // configuration of gesvdj
    constexpr double tol = 1.e-7;
    constexpr int max_sweeps = 15;

    // compute eigenvectors
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
    // econ = 1 for economy size
    constexpr int econ = 0;

    // numerical results of gesvdj
    double residual = 0.0;

    int executed_sweeps = 0;

    std::cout << "example of gesvdj \n";
    printf("tol = %E, default value is machine zero \n", tol);
    printf("max. sweeps = %d, default value is 100\n", max_sweeps);
    printf("econ = %d \n", econ);

    std::cout << "A = (matlab base-1)\n";
    printMatrix(rows_A, cols_A, A.data(), rows_A, "A");
    std::cout << "=====\n";

    // step 1: create cusolver handle, bind a stream
    check(hipsolverDnCreate(&cusolverH));
    check(hipblasCreate(&cublasH));
    check(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    check(hipsolverSetStream(cusolverH, stream));

    // step 2: configuration of gesvdj
    check(hipsolverDnCreateGesvdjInfo(&gesvdj_params));
    // default value of tolerance is machine zero
    check(hipsolverDnXgesvdjSetTolerance(gesvdj_params, tol));
    // default value of max. sweeps is 100
    check(hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, max_sweeps));

    // step 3: copy to device
    check(hipMalloc((void**)&d_A, sizeof(double) * rows_A * cols_A));
    check(hipMalloc((void**)&d_S, sizeof(double) * cols_A));
    check(hipMalloc((void**)&d_U, sizeof(double) * rows_A * rows_A));
    check(hipMalloc((void**)&d_V, sizeof(double) * cols_A * cols_A));
    check(hipMalloc((void**)&d_info, sizeof(int)));

    check(hipMemcpy(d_A, A.data(), sizeof(double) * rows_A * cols_A, hipMemcpyHostToDevice));

    std::cout << "random number generator\n";

    double* d_random = nullptr;
    int rows_random = cols_A;
    int cols_random = modes;
    check(hipMalloc((void**)&d_random, rows_random * cols_random * sizeof(double)));

    hiprandGenerator_t gen;
    /* Create pseudo-random number generator */
    check(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

    /* Set seed */
    check(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

    /* Generate n floats on device */
    check(hiprandGenerateUniformDouble(gen, d_random, rows_random * cols_random));

    std::cout << "compute approximate basis\n";

    double alpha = 1.0;
    double beta = 0.0;

    gpu_matrix g_basis(rows_A, modes);
    std::cout << "/* message */" << g_basis.rows() << '\n';
    std::cout << "/* message */" << g_basis.columns() << '\n';
    std::cout << "/* ------------------------------------------- */" << '\n';

    double* d_basis = nullptr;
    int rows_basis = rows_A;
    int cols_basis = modes;
    check(hipMalloc((void**)&d_basis, sizeof(double) * rows_basis * cols_basis));

    check(cublasDgemm(cublasH,
                      HIPBLAS_OP_N,
                      HIPBLAS_OP_N,
                      rows_basis,
                      cols_basis,
                      cols_A,
                      &alpha,
                      d_A,
                      rows_A,
                      d_random,
                      rows_random,
                      &beta,
                      d_basis,
                      rows_basis));

    std::cout << "orthonormalise the approximate basis\n";
    // perform QR decomposition on d_basis
    int lwork = 0;
    check(hipsolverDnDgeqrf_bufferSize(cusolverH, rows_basis, cols_basis, d_basis, rows_basis, &lwork));
    check(hipMalloc((void**)&d_work, sizeof(double) * lwork));

    double* d_tau = nullptr;
    check(hipMalloc((void**)&d_tau, sizeof(double) * cols_A));

    check(hipsolverDnDgeqrf(cusolverH,
                           rows_basis,
                           cols_basis,
                           d_basis,
                           rows_basis,
                           d_tau,
                           d_work,
                           lwork,
                           d_info));
    // calculate Q
    check(hipsolverDnDorgqr_bufferSize(cusolverH,
                                      rows_basis,
                                      cols_basis,
                                      cols_basis,
                                      d_basis,
                                      rows_basis,
                                      d_tau,
                                      &lwork));

    check(hipsolverDnDorgqr(cusolverH,
                           rows_basis,
                           cols_basis,
                           cols_basis,
                           d_basis,
                           rows_basis,
                           d_tau,
                           d_work,
                           lwork,
                           d_info));

    std::cout << "restrict A to its "
                 "approximate basis\n";

    double* d_A_reduced = nullptr;
    int rows_A_reduced = modes;
    int cols_A_reduced = cols_A;
    check(hipMalloc((void**)&d_A_reduced, sizeof(double) * rows_A_reduced * cols_A_reduced));

    check(cublasDgemm(cublasH,
                      HIPBLAS_OP_T,
                      HIPBLAS_OP_N,
                      rows_A_reduced,
                      cols_A_reduced,
                      rows_A,
                      &alpha,
                      d_basis,
                      rows_basis,
                      d_A,
                      rows_A,
                      &beta,
                      d_A_reduced,
                      rows_A_reduced));

    std::cout << "randomised svd solver\n";

    double* d_U_reduced = nullptr;
    int rows_U_reduced = rows_A_reduced;
    int cols_U_reduced = rows_A_reduced;
    check(hipMalloc((void**)&d_U_reduced, sizeof(double) * rows_U_reduced * cols_U_reduced));

    // size of workspace
    check(hipsolverDnDgesvdj_bufferSize(cusolverH,
                                       jobz,
                                       econ,
                                       rows_A_reduced,
                                       cols_A_reduced,
                                       d_A_reduced,
                                       rows_A_reduced,
                                       d_S,
                                       d_U_reduced,
                                       rows_U_reduced,
                                       d_V,
                                       cols_A_reduced,
                                       &lwork,
                                       gesvdj_params));

    check(hipMalloc((void**)&d_work, sizeof(double) * lwork));

    check(hipsolverDnDgesvdj(cusolverH,
                            jobz,
                            econ,
                            rows_A_reduced,
                            cols_A_reduced,
                            d_A_reduced,
                            rows_A_reduced,
                            d_S,
                            d_U_reduced,
                            rows_U_reduced,
                            d_V,
                            cols_A_reduced,
                            d_work,
                            lwork,
                            d_info,
                            gesvdj_params));

    check(cublasDgemm(cublasH,
                      HIPBLAS_OP_N,
                      HIPBLAS_OP_N,
                      rows_A,
                      rows_A,
                      cols_basis,
                      &alpha,
                      d_basis,
                      rows_basis,
                      d_U_reduced,
                      rows_U_reduced,
                      &beta,
                      d_U,
                      rows_A));

    std::cout << "compare the results & ...\n";
    // TODO

    std::cout << "svd solver\n";
    // step 4: query workspace of SVD

    // size of workspace
    check(hipsolverDnDgesvdj_bufferSize(cusolverH,
                                       jobz, // CUSOLVER_EIG_MODE_NOVECTOR:
                                             // compute singular values only
                                       // CUSOLVER_EIG_MODE_VECTOR: compute
                                       // singular value and singular vectors
                                       econ,   // econ = 1 for economy size
                                       rows_A, // nubmer of rows of A, 0 <= m
                                       cols_A, // number of columns of A, 0 <=
                                               // n
                                       d_A,    // m-by-n
                                       rows_A, // leading dimension of A
                                       d_S,    // min(m,n)
                                               // the singular values in
                                               // descending order
                                       d_U,    // m-by-m if econ = 0
                                               // m-by-min(m,n) if econ = 1
                                       rows_A, // leading dimension of U, ldu
                                               // >= max(1,m)
                                       d_V,    // n-by-n if econ = 0
                                               // n-by-min(m,n) if econ = 1
                                       rows_A, // leading dimension of V, ldv
                                               // >= max(1,n)
                                       &lwork,
                                       gesvdj_params));

    check(hipMalloc((void**)&d_work, sizeof(double) * lwork));

    // step 5: compute SVD
    check(hipsolverDnDgesvdj(cusolverH,
                            jobz, // CUSOLVER_EIG_MODE_NOVECTOR:
                                  // compute singular values only
                            // CUSOLVER_EIG_MODE_VECTOR: compute
                            // singular value and singular vectors
                            econ,   // econ = 1 for economy size
                            rows_A, // nubmer of rows of A, 0 <= m
                            cols_A, // number of columns of A, 0 <=
                                    // n
                            d_A,    // m-by-n
                            rows_A, // leading dimension of A
                            d_S,    // min(m,n)
                                    // the singular values in
                                    // descending order
                            d_U,    // m-by-m if econ = 0
                                    // m-by-min(m,n) if econ = 1
                            rows_A, // leading dimension of U, ldu
                                    // >= max(1,m)
                            d_V,    // n-by-n if econ = 0
                                    // n-by-min(m,n) if econ = 1
                            rows_A, // leading dimension of V, ldv
                                    // >= max(1,n)
                            d_work,
                            lwork,
                            d_info,
                            gesvdj_params));

    check(hipDeviceSynchronize());

    check(hipMemcpy(U.data(), d_U, sizeof(double) * rows_A * rows_A, hipMemcpyDeviceToHost));
    check(hipMemcpy(V.data(), d_V, sizeof(double) * cols_A * cols_A, hipMemcpyDeviceToHost));
    check(hipMemcpy(S.data(), d_S, sizeof(double) * cols_A, hipMemcpyDeviceToHost));
    check(hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost));

    check(hipDeviceSynchronize());

    if (info == 0)
    {
        std::cout << "gesvdj converges\n";
    }
    else if (0 > info)
    {
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }
    else
    {
        printf("WARNING: info = %d : gesvdj did "
               "not converge \n",
               info);
    }

    std::cout << "S = singular values (matlab "
                 "base-1)\n";
    printMatrix(cols_A, 1, S.data(), rows_A, "S");
    std::cout << "=====\n";

    std::cout << "U = left singular vectors "
                 "(matlab base-1)\n";
    printMatrix(rows_A, rows_A, U.data(), rows_A, "U");
    std::cout << "=====\n";

    std::cout << "V = right singular vectors "
                 "(matlab base-1)\n";
    printMatrix(cols_A, cols_A, V.data(), rows_A, "V");
    std::cout << "=====\n";

    /* step 6: measure error of singular value
     */
    double ds_sup = 0.0;
    for (int j = 0; j < cols_A; j++)
    {
        ds_sup = std::max(ds_sup, std::abs(S[j] - S_exact[j]));
    }
    printf("|S - S_exact|_sup = %E \n", ds_sup);

    check(hipsolverDnXgesvdjGetSweeps(cusolverH, gesvdj_params, &executed_sweeps));

    check(hipsolverDnXgesvdjGetResidual(cusolverH, gesvdj_params, &residual));

    printf("residual |A - U*S*V**H|_F = %E \n", residual);
    printf("number of executed sweeps = %d \n", executed_sweeps);

    if (d_A) hipFree(d_A);
    if (d_S) hipFree(d_S);
    if (d_U) hipFree(d_U);
    if (d_V) hipFree(d_V);
    if (d_info) hipFree(d_info);
    if (d_work) hipFree(d_work);

    if (cusolverH) hipsolverDnDestroy(cusolverH);
    if (stream) hipStreamDestroy(stream);
    if (gesvdj_params) hipsolverDnDestroyGesvdjInfo(gesvdj_params);
    if (cublasH) hipblasDestroy(cublasH);

    hipDeviceReset();

    return 0;
}
